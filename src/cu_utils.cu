#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hipblas.h"
#include "headers/cu_utils.h"
#include "hipsparse.h"

using namespace std;

#define threadsPerBlock 1024



// CUDA scaling

template<class T>
__global__ void cuda_Tscal_kernel(int vectorSize, T alpha, T *vector)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < vectorSize)
    vector[idx] = alpha * vector[idx];
}

void cudaTscal(int vectorSize, float *alpha, float *vector)
{
   int numBlocks = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;

   cuda_Tscal_kernel<float><<< numBlocks, threadsPerBlock>>>( vectorSize, *alpha, vector);
   hipDeviceSynchronize();
}

void cudaTscal(int vectorSize, double *alpha, double *vector)
{
   int numBlocks = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;

   cuda_Tscal_kernel<double><<< numBlocks, threadsPerBlock>>>( vectorSize, *alpha, vector);
   hipDeviceSynchronize();
}

// cuBLAS scaling

hipblasStatus_t cublasTscal(hipblasHandle_t handle, int n, const float *alpha, float *x, int incx)
{
    return hipblasSscal(handle, n, alpha, x, incx);
}


hipblasStatus_t cublasTscal(hipblasHandle_t handle, int n, const double *alpha, double *x, int incx)
{
    return hipblasDscal(handle, n, alpha, x, incx);
}

// cuBLAS dot product 

hipblasStatus_t cublasTdot (hipblasHandle_t handle, int n, const float *x, int incx, const float *y, int incy, float *result)
{
  return hipblasSdot (handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublasTdot (hipblasHandle_t handle, int n, const double *x, int incx, const double *y, int incy, double *result)
{
  return hipblasDdot (handle, n, x, incx, y, incy, result);
}


// cuBLAS axpy

hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const float *alpha, const float *x, int incx, float *y, int incy)
{
  return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y, int incy)
{
  return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

// cuSparse mv  

hipsparseStatus_t cusparseTcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const float *alpha, const hipsparseMatDescr_t descrA, const float *csrValA, const int *csrRowPtrA, const int *csrColIndA, const float *x, const float *beta, float *y)
{
  return hipsparseScsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, beta, y);
}

hipsparseStatus_t cusparseTcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const double *alpha, const hipsparseMatDescr_t descrA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, const double *x, const double *beta, double *y)
{
  return hipsparseDcsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, beta, y);
}
