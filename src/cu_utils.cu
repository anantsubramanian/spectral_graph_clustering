#include <stdio.h>
#include "hipblas.h"
/*#include <hipblas.h>*/


/**
 * Computes a*x + y for two dense vectors x and y.
 */

extern void cu_daxpy( float *result, float a, float *x, float *y, int vec_size )
{
  hipSetDevice(0);

  /*int i,j;*/
  float *devPtrX , *devPtrY;

  //timer stuff
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*printf ("\nvecX\n");*/
  /*for(i=0;i<vec_size;i++) printf("%f ",x[i]);*/
  /*printf ("\nvecY\n");*/
  /*for(i=0;i<vec_size;i++) printf("%f ",y[i]);*/

  /*printf ("\n\n");*/

  hipMalloc( (void**) &devPtrX, vec_size * sizeof(float));	//matrix A
  hipMalloc( (void**) &devPtrY, vec_size * sizeof(float));		//vector B

  // transfer host data to device
  hipblasSetVector( vec_size, sizeof(float), x, 1, devPtrX, 1);
  hipblasSetVector( vec_size, sizeof(float), y, 1, devPtrY, 1);

  //start timer 
  hipEventRecord(start, 0);

  // do saxpy
  /*hipblasStatus_t stat;*/
  hipblasHandle_t handle;

  hipblasCreate(&handle);

  hipblasSaxpy(handle, vec_size, &a, devPtrX, 1, devPtrY, 1);

  /*// block until the device has completed*/
  hipDeviceSynchronize();

  //end timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);		

  hipblasGetVector (vec_size, sizeof(float), devPtrY, 1, y, 1);

  /*for (j = 0; j < vec_size; j++) printf ("%7.0f", y[j]);//IDX2C(j,1,M)]);*/
  /*printf("\n\ntime = %f\n\n",elapsedTime);*/

  hipEventDestroy(start);
  hipEventDestroy(stop);

  result = y;

}
