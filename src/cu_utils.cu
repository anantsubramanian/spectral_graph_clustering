#include "hip/hip_runtime.h"
#include <cstdio>
#include "hipblas.h"
#include "headers/cu_utils.h"
#include "hipsparse.h"

using namespace std;

#define threadsPerBlock 1024 // max threads on Tesla K40 GPU

// CUDA scaling kernel

template <class T>
__global__ void cuda_Tscal_kernel (int vectorSize, T alpha, T *vector)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < vectorSize)
    vector[idx] = alpha * vector[idx];
}

// CUDA - Vector scaling wrapper
// vector = alpha * vector

template <class T>
void cudaTscal (int vectorSize, T *alpha, T *vector)
{
  int numBlocks = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;

  cuda_Tscal_kernel<float><<< numBlocks, threadsPerBlock>>>( vectorSize, *alpha, vector);
  hipDeviceSynchronize();
}


// cuBLAS - Vector scaling wrappers
// x = alpha * x; incx - stride, n - number of elements

hipblasStatus_t cublasTscal (
    hipblasHandle_t handle, int n, const float *alpha, float *x,
    int incx )
{
  return hipblasSscal(handle, n, alpha, x, incx);
}


hipblasStatus_t cublasTscal (
    hipblasHandle_t handle, int n, const double *alpha, double *x,
    int incx )
{
  return hipblasDscal(handle, n, alpha, x, incx);
}


// cuBLAS - Dot product wrapper
// result = x.y; incx and incy - strides, n - number of elements

hipblasStatus_t cublasTdot (
    hipblasHandle_t handle, int n, const float *x, int incx, const float *y, int incy,
    float *result )
{
  return hipblasSdot (handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublasTdot (
    hipblasHandle_t handle, int n, const double *x, int incx, const double *y, int incy,
    double *result )
{
  return hipblasDdot (handle, n, x, incx, y, incy, result);
}


// cuBLAS - axpy wrapper
// y = alpha * x + y; incx and incy - strides, n - number of elements

hipblasStatus_t cublasTaxpy (
    hipblasHandle_t handle, int n, const float *alpha, const float *x, int incx, float *y,
    int incy )
{
  return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasTaxpy (
    hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y,
    int incy )
{
  return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}


// cuSparse - Sparse CSR Matrix Vector Multiplication wrapper
// y = alpha * matrixA * x + beta * y; m and n - dimensions, nnz - number of non-zero elements

hipsparseStatus_t cusparseTcsrmv(
    hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz,
    const float *alpha, const hipsparseMatDescr_t descrA, const float *csrValA,
    const int *csrRowPtrA, const int *csrColIndA, const float *x, const float *beta,
    float *y )
{
  return hipsparseScsrmv(
      handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA,
      csrColIndA, x, beta, y);
}

hipsparseStatus_t cusparseTcsrmv(
    hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz,
    const double *alpha, const hipsparseMatDescr_t descrA, const double *csrValA,
    const int *csrRowPtrA, const int *csrColIndA, const double *x,
    const double *beta, double *y )
{
  return hipsparseDcsrmv(
      handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA,
      csrColIndA, x, beta, y);
}


// cuSparse - Convert CSR/CSC matrix a CSC/CSR matrix resp.

hipsparseStatus_t cusparseTcsr2csc(
    hipsparseHandle_t handle, int m, int n, int nnz, const float *csrVal,
    const int *csrRowPtr, const int *csrColInd, float *cscVal, int *cscRowInd,
    int *cscColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase)
{
  return hipsparseScsr2csc(
      handle, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscRowInd,
      cscColPtr, copyValues, idxBase);
}

hipsparseStatus_t cusparseTcsr2csc(
    hipsparseHandle_t handle, int m, int n, int nnz, const double *csrVal,
    const int *csrRowPtr, const int *csrColInd, double *cscVal, int *cscRowInd,
    int *cscColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase)
{
  return hipsparseDcsr2csc(
      handle, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscRowInd,
      cscColPtr, copyValues, idxBase);
}

