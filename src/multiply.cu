#include <stdio.h>
#include <hipblas.h>

extern void call_me_maybe( float *vecA, float *vecB, int N, int M)
{
  hipSetDevice(0);

  int i,j;
  float *devPtrA , *devPtrB , *devPtrC;

  //timer stuff
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  printf ("\nvecA\n");
  for(i=0;i<N;i++) printf("%f ",vecA[i]);
  printf ("\nvecB\n");
  for(i=0;i<N;i++) printf("%f ",vecB[i]);

  printf ("\n\n");


  cublasInit(); // initilization of CUDA application
  cublasAlloc( N, sizeof(float), (void**) &devPtrA);	//matrix A
  cublasAlloc( N, sizeof(float), (void**) &devPtrB);		//vector B
  cublasAlloc( N, sizeof(float), (void**) &devPtrC);		//vector C


  // transfer host data to device
  hipblasSetVector( N, sizeof(float), vecA, 1, devPtrA, 1);
  hipblasSetVector( N, sizeof(float), vecB, 1, devPtrB, 1);

  // compute C = A*B in device
  float alpha = 1.0;
  float beta  = 0.0;

  //start timer 
  hipEventRecord(start, 0);
  /*hipblasSgemv('N', M, N, alpha, devPtrA, M, devPtrB, 1, beta, devPtrC, 1);*/
  /*hipblasSdot(N, devPtrA, 1, devPtrB, 1);*/

    hipblasHandle_t handle;
    int n = 5;
  /*hipblasStatus_t hipblasSdot (handle, n, devPtrA, 1, devPtrB, 1, devPtrC);*/

  // block until the device has completed
  hipDeviceSynchronize();

  //end timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);		

  hipblasGetVector (N, sizeof(float), devPtrB, 1, vecB, 1);


  for (j = 0; j < N; j++) printf ("%7.0f", vecB[j]);//IDX2C(j,1,M)]);
  printf("\n\ntime = %f\n\n",elapsedTime);

  cublasShutdown(); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
